#include "hip/hip_runtime.h"
#include "calculate.hpp"

#include <cassert>
#include <cmath>
#include <fstream>
#include <stdint.h>

#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "variables.hpp"

const size_t BLOCKSIZE = 256;
// const double START_RE   = -1.75f;
// const double START_IM   = -1.75f;
const double NORM_LIMIT = 1000;

struct Complex {
  double p_re, p_im;
  __device__ Complex(const double re, const double im) : p_re(re), p_im(im) {
  }
  __device__ Complex operator*(const Complex& a) const {
    return Complex(p_re * a.p_re - p_im * a.p_im, p_re * a.p_im + p_im * a.p_re);
  }
  __device__ Complex operator+(const Complex& a) const {
    return Complex(p_re + a.p_re, p_im + a.p_im);
  }
  __device__ double squaredAbs() const {
    return p_re * p_re + p_im * p_im;
  }
};

__device__ static Complex function(const Complex& z) {
  return z * z + Complex(-0.78, -0.18);
}

/*
__global__ static void calculatePixelsGPU(
  __int16_t* pixels, const size_t imageSize, const size_t width, const double step,
  const size_t max_iter) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < imageSize) {
    int idx             = id % width;
    int idy             = id / width;
    __int16_t iteration = 0;
    Complex z           = Complex(START_RE + idx * step, START_IM + idy * step);
    do {
      z = function(z);
      ++iteration;
    } while (iteration < max_iter && z.squaredAbs() < NORM_LIMIT);
    pixels[width * idy + idx] = iteration;
  }
}
*/

/*
// calculate just half of the pixels due to symmetrie
void julia_fatouCUDA(const char* filename, const double step, const size_t max_iter) {
  const size_t width       = std::abs(double(START_RE * 2 / step));
  const size_t half_height = std::abs(double(START_IM / step));
  const size_t imageSize   = half_height * width;

  __int16_t* pixels;
  hipHostAlloc((void**) &pixels, imageSize * sizeof(__int16_t), 0);

  dim3 blockDim(BLOCKSIZE);
  dim3 gridDim(std::ceil(imageSize / (float) BLOCKSIZE));

  __int16_t* cudaPixels;
  hipMalloc((void**) &cudaPixels, imageSize * sizeof(__int16_t));

  calculatePixelsGPU<<<gridDim, blockDim>>>(cudaPixels, imageSize, width, step, max_iter);

  hipMemcpy(pixels, cudaPixels, imageSize * sizeof(__int16_t), hipMemcpyDeviceToHost);

  hipFree(cudaPixels);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  std::ofstream myfile(filename, std::ios::binary);
  myfile.write((char*) &width, sizeof(width));
  myfile.write((char*) &half_height, sizeof(half_height));
  myfile.write((char*) pixels, imageSize * sizeof(__int16_t));
  assert(myfile.fail() == 0 && "Could not write correctly!");
  myfile.close();

  hipHostFree(pixels);
}
*/

// iterate the function and calculate the color
__global__ static void calculatePixelsGPU(
  Byte* cudaPixels, const unsigned int imageSize, const unsigned int width, const float step,
  const unsigned int max_iter, const float startRe, const float startIm, const float red,
  const float green, const float blue) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < imageSize) {
    int idx             = id % width;
    int idy             = id / width;
    __int16_t iteration = 0;
    Complex z           = Complex(startRe + idx * step, startIm + idy * step);
    do {
      z = function(z);
      ++iteration;
    } while (iteration < max_iter && z.squaredAbs() < NORM_LIMIT);

    // calculate colors and write
    cudaPixels[3 * (width * idy + idx)]     = std::round(red * iteration);
    cudaPixels[3 * (width * idy + idx) + 1] = std::round(green * iteration);
    cudaPixels[3 * (width * idy + idx) + 2] = std::round(blue * iteration);
  }
}

using namespace functionParameters;
// do calculation adjusted to the the displaying
void juliaFatouCUDA(Byte* textureImg) {
  const unsigned int imageSize = mainWindow::WIDTH * mainWindow::HEIGHT;

  // set up grid
  dim3 blockDim(BLOCKSIZE);
  dim3 gridDim(std::ceil(imageSize / (float) BLOCKSIZE));

  // allocate GPU memory
  Byte* cudaPixels;
  hipMalloc((void**) &cudaPixels, imageSize * universal::RGB_COLORS);

  // do computation
  calculatePixelsGPU<<<gridDim, blockDim>>>(
    cudaPixels, imageSize, mainWindow::WIDTH, STEP, MAX_ITER, RE_START, IM_START, RED, GREEN, BLUE);

  // copy memory from GRAM to RAM
  hipMemcpy(textureImg, cudaPixels, imageSize * universal::RGB_COLORS, hipMemcpyDeviceToHost);

  // free the allocated memory on GPU
  hipFree(cudaPixels);

  // print status cuda message
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  /*
  // DEBUG
  for(unsigned int i =0; i<imageSize * universal::RGB_COLORS; ++i) {
    textureImg[i] = 0.3;
  }
  */
}
