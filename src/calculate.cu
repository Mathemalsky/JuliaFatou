#include "hip/hip_runtime.h"
#include "calculate.hpp"

#include <cassert>
#include <cmath>
#include <fstream>
#include <stdint.h>

#include <hip/hip_runtime.h>

#include <iostream>

struct Complex {
  double p_re, p_im;
  __device__ Complex(const double re, const double im) : p_re(re), p_im(im) {
  }
  __device__ Complex operator*(const Complex& a) const {
    return Complex(p_re * a.p_re - p_im * a.p_im, p_re * a.p_im + p_im * a.p_re);
  }
  __device__ Complex operator+(const Complex& a) const {
    return Complex(p_re + a.p_re, p_im + a.p_im);
  }
  __device__ double squaredAbs() const {
    return p_re * p_re + p_im * p_im;
  }
};

__device__ static Complex function(const Complex& z) {
  return z * z + Complex(0.2, -0.6);
}

__global__ void calculatePixelsGPU(__int16_t* pixels, const double step, const size_t max_iter) {
  int i            = blockIdx.x;
  int j            = blockIdx.y;
  size_t iteration = 0;
  Complex z        = Complex(-1.75f + i * step, -1.75f + j * step);
  do {
    z = function(z);
    ++iteration;
  } while (iteration < max_iter && z.squaredAbs() < 4);
  pixels[gridDim.x * j + i] = iteration;
}

// calculate just half of the pixels due to symmetrie
void julia_fatouCUDA(const char* filename, const double step, const size_t max_iter) {
  const double start_re = -1.75f;
  const double start_im = -1.75f;
  // const double norm_limit = 4;

  const size_t width       = std::abs(double(start_re * 2 / step));
  const size_t half_height = std::abs(double(start_im / step));
  const size_t imageSize   = half_height * width;
  //__int16_t* pixels        = (__int16_t*) malloc(imageSize * sizeof(__int16_t));
  __int16_t* pixels;
  hipHostAlloc((void**) &pixels, imageSize * sizeof(__int16_t), 0);

  dim3 grid(width, half_height);

  __int16_t* cudaPixels;
  hipMalloc((void**) &cudaPixels, imageSize * sizeof(__int16_t));

  calculatePixelsGPU<<<grid, 1>>>(cudaPixels, step, max_iter);

  hipMemcpy(pixels, cudaPixels, imageSize * sizeof(__int16_t), hipMemcpyDeviceToHost);

  hipFree(cudaPixels);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  std::ofstream myfile(filename, std::ios::binary);
  myfile.write((char*) &width, sizeof(width));
  myfile.write((char*) &half_height, sizeof(half_height));
  myfile.write((char*) pixels, imageSize * sizeof(__int16_t));
  assert(myfile.fail() == 0 && "Could not write correctly!");
  myfile.close();

  // free(pixels);
  hipHostFree(pixels);
}
