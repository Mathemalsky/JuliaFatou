#include "hip/hip_runtime.h"
#include "calculate.hpp"

#include <cassert>
#include <cmath>
#include <fstream>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "types.hpp"
#include "variables.hpp"

#include <iostream>

struct Complex {
  double p_re, p_im;
  __device__ Complex(const double re, const double im) : p_re(re), p_im(im) {
  }
  __device__ Complex operator*(const Complex& a) const {
    return Complex(p_re * a.p_re - p_im * a.p_im, p_re * a.p_im + p_im * a.p_re);
  }
  __device__ Complex operator/(const Complex& a) const {
    const double r2 = a.squaredAbs();
    return Complex((p_re * a.p_re + p_im * a.p_im) / r2, (p_re * a.p_im - p_im * a.p_re) / r2);
  }
  __device__ Complex operator+(const Complex& a) const {
    return Complex(p_re + a.p_re, p_im + a.p_im);
  }
  __device__ Complex operator-(const Complex& a) const {
    return Complex(p_re - a.p_re, p_im - a.p_im);
  }
  __device__ double squaredAbs() const {
    return p_re * p_re + p_im * p_im;
  }
};

__device__ static Complex function(const Complex& z, const float reOffset, const float imOffset) {
  return z * z + Complex(reOffset, imOffset);
}

// iterate the function and calculate the color
__global__ static void calculatePixelsGPU(
  Byte* cudaPixels, const unsigned int imageSize, const unsigned int width, const float step,
  const unsigned int max_iter, const float startRe, const float startIm, const float reOffset, const float imOffset,
  const RGB dCol, const RGB cCol, const bool check_conv) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < imageSize) {
    int idx             = id % width;
    int idy             = id / width;
    __int16_t iteration = 0;
    Complex z           = Complex(startRe + idx * step, startIm + idy * step);
    Complex z_prev(0.0, 0.0);
    do {
      z_prev = z;
      z      = function(z, reOffset, imOffset);
      ++iteration;
      if (check_conv && (z / z_prev - Complex(1, 0)).squaredAbs() < functionParameters::CONVERGENCE_LIMIT) {
        iteration = -iteration;
        break;  // escapes the while loop
      }
    } while (iteration < max_iter && z.squaredAbs() < functionParameters::NORM_LIMIT);

    // calculate colors and write
    if (iteration > 0) {
      cudaPixels[3 * (width * idy + idx)]     = std::round((dCol.red * iteration * universal::MAX_BYTE) / max_iter);
      cudaPixels[3 * (width * idy + idx) + 1] = std::round((dCol.green * iteration * universal::MAX_BYTE) / max_iter);
      cudaPixels[3 * (width * idy + idx) + 2] = std::round((dCol.blue * iteration * universal::MAX_BYTE) / max_iter);
    }
    else {
      iteration                               = -iteration;
      cudaPixels[3 * (width * idy + idx)]     = std::round((cCol.red * iteration * universal::MAX_BYTE) / max_iter);
      cudaPixels[3 * (width * idy + idx) + 1] = std::round((cCol.green * iteration * universal::MAX_BYTE) / max_iter);
      cudaPixels[3 * (width * idy + idx) + 2] = std::round((cCol.blue * iteration * universal::MAX_BYTE) / max_iter);
    }
  }
}

void* allocateGraphicsMemory() {
  void* cudaPixels;
  hipMalloc((void**) &cudaPixels, mainWindow::MAX_WIDTH_X_HEIGHT * universal::RGB_COLORS);
  return cudaPixels;
}

void freeGraphicsMemory(void* cudaPixels) {
  hipFree(cudaPixels);
}

using namespace functionParameters;
// do calculation adjusted to the the displaying
void juliaFatouCUDA(Byte* textureImg, void* cudaPixels) {
  const unsigned int imageSize = mainWindow::WIDTH * mainWindow::HEIGHT;

  // set up grid
  dim3 blockDim(settingsGPU::BLOCKSIZE, 1, 1);
  dim3 gridDim(std::ceil(imageSize / (float) settingsGPU::BLOCKSIZE), 1, 1);

  // do computation
  calculatePixelsGPU<<<gridDim, blockDim>>>(
    (Byte*) cudaPixels, imageSize, mainWindow::WIDTH, STEP, MAX_ITER, RE_START, IM_START, RE_OFFSET, IM_OFFSET,
    RGB{D_RED, D_GREEN, D_BLUE}, RGB{C_RED, C_GREEN, C_BLUE}, imGuiWindow::CALC_CONVERGENCE);

  // copy memory from GRAM to RAM
  hipMemcpy(textureImg, cudaPixels, imageSize * universal::RGB_COLORS, hipMemcpyDeviceToHost);
}

void singleBigFrame(Byte* pixels) {
  const unsigned int size = SCREENSHOT_WIDTH * SCREENSHOT_HEIGHT;

  // compute different start end end for picture
  const double step =
    STEP * std::max((double) mainWindow::WIDTH / SCREENSHOT_WIDTH, (double) mainWindow::HEIGHT / SCREENSHOT_HEIGHT);
  const double reStart = RE_START + STEP * mainWindow::WIDTH / 2.0 - step * SCREENSHOT_WIDTH / 2.0;
  const double imStart = IM_START + STEP * mainWindow::HEIGHT / 2.0 - step * SCREENSHOT_HEIGHT / 2.0;

  // set up grid
  dim3 blockDim(settingsGPU::BLOCKSIZE, 1, 1);
  dim3 gridDim(std::ceil(size / (float) settingsGPU::BLOCKSIZE), 1, 1);

  // allocate graphics memory
  Byte* cudaPixels;
  hipMalloc((void**) &cudaPixels, size * universal::RGB_COLORS * sizeof(Byte));

  // compute image for the screenshot
  calculatePixelsGPU<<<gridDim, blockDim>>>(
    cudaPixels, size, SCREENSHOT_WIDTH, step, MAX_ITER, reStart, imStart, RE_OFFSET, IM_OFFSET,
    RGB{D_RED, D_GREEN, D_BLUE}, RGB{C_RED, C_GREEN, C_BLUE}, imGuiWindow::CALC_CONVERGENCE);

  hipMemcpy(pixels, cudaPixels, size * universal::RGB_COLORS, hipMemcpyDeviceToHost);

  // keep track of errors in cuda functions
  assert(hipGetErrorString(hipGetLastError()) == "no error");

  hipFree(cudaPixels);
}
