#include "hip/hip_runtime.h"
#include "calculate.hpp"

#include <cassert>
#include <cmath>
#include <fstream>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "constants.hpp"
#include "variables.hpp"

#include <iostream>

struct Complex {
  double p_re, p_im;
  __device__ Complex(const double re, const double im) : p_re(re), p_im(im) {
  }
  __device__ Complex operator*(const Complex& a) const {
    return Complex(p_re * a.p_re - p_im * a.p_im, p_re * a.p_im + p_im * a.p_re);
  }
  __device__ Complex operator+(const Complex& a) const {
    return Complex(p_re + a.p_re, p_im + a.p_im);
  }
  __device__ double squaredAbs() const {
    return p_re * p_re + p_im * p_im;
  }
};

__device__ static Complex function(const Complex& z) {
  return z * z + Complex(-0.78, -0.18);
}

// iterate the function and calculate the color
__global__ static void calculatePixelsGPU(
  Byte* cudaPixels, const unsigned int imageSize, const unsigned int width, const float step,
  const unsigned int max_iter, const float startRe, const float startIm, const float red, const float green,
  const float blue) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < imageSize) {
    int idx             = id % width;
    int idy             = id / width;
    __int16_t iteration = 0;
    Complex z           = Complex(startRe + idx * step, startIm + idy * step);
    do {
      z = function(z);
      ++iteration;
    } while (iteration < max_iter && z.squaredAbs() < functionParameters::NORM_LIMIT);

    // calculate colors and write
    cudaPixels[3 * (width * idy + idx)]     = std::round((red * iteration * universal::MAX_BYTE) / max_iter);
    cudaPixels[3 * (width * idy + idx) + 1] = std::round((green * iteration * universal::MAX_BYTE) / max_iter);
    cudaPixels[3 * (width * idy + idx) + 2] = std::round((blue * iteration * universal::MAX_BYTE) / max_iter);
  }
}

void* allocateGraphicsMemory() {
  void* cudaPixels;
  const unsigned int imageSize = mainWindow::WIDTH * mainWindow::HEIGHT;
  hipMalloc((void**) &cudaPixels, imageSize * universal::RGB_COLORS);
  return cudaPixels;
}

void freeGraphicsMemory(void* cudaPixels) {
  hipFree(cudaPixels);
}

using namespace functionParameters;
// do calculation adjusted to the the displaying
void juliaFatouCUDA(Byte* textureImg, void* cudaPixels) {
  const unsigned int imageSize = mainWindow::WIDTH * mainWindow::HEIGHT;

  // set up grid
  dim3 blockDim(settingsGPU::BLOCKSIZE, 1, 1);
  dim3 gridDim(std::ceil(imageSize / (float) settingsGPU::BLOCKSIZE), 1, 1);

  // do computation
  calculatePixelsGPU<<<gridDim, blockDim>>>(
    (Byte*) cudaPixels, imageSize, mainWindow::WIDTH, STEP, MAX_ITER, RE_START, IM_START, RED, GREEN, BLUE);

  // copy memory from GRAM to RAM
  hipMemcpy(textureImg, cudaPixels, imageSize * universal::RGB_COLORS, hipMemcpyDeviceToHost);
}

void singleBigFrame(Byte* pixels) {
  const unsigned int size = SCREENSHOT_WIDTH * SCREENSHOT_HEIGHT;

  // compute different start end end for picture
  const double step =
    STEP * std::max((double) mainWindow::WIDTH / SCREENSHOT_WIDTH, (double) mainWindow::HEIGHT / SCREENSHOT_HEIGHT);
  const double reStart = RE_START + STEP * mainWindow::WIDTH / 2.0 - step * SCREENSHOT_WIDTH / 2.0;
  const double imStart = IM_START + STEP * mainWindow::HEIGHT / 2.0 - step * SCREENSHOT_HEIGHT / 2.0;

  // set up grid
  dim3 blockDim(settingsGPU::BLOCKSIZE, 1, 1);
  dim3 gridDim(std::ceil(size / (float) settingsGPU::BLOCKSIZE), 1, 1);

  // allocate graphics memory
  Byte* cudaPixels;
  hipMalloc((void**) &cudaPixels, size * universal::RGB_COLORS * sizeof(Byte));

  // compute image for the screenshot
  calculatePixelsGPU<<<gridDim, blockDim>>>(
    cudaPixels, size, SCREENSHOT_WIDTH, step, MAX_ITER, reStart, imStart, RED, GREEN, BLUE);

  hipMemcpy(pixels, cudaPixels, size * universal::RGB_COLORS, hipMemcpyDeviceToHost);

  // keep track of errors in cuda functions
  assert(hipGetErrorString(hipGetLastError()) == "no error");

  hipFree(cudaPixels);
}
